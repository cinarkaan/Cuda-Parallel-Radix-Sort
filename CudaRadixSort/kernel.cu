#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define blockSize 128

__global__ void kernelCount(int* G_arr,int *G_Count,int dataSize,int digit) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < dataSize)
        atomicAdd(&(G_Count[(G_arr[idx] / digit) % 10]), 1);
    else
        return;
}

__global__ void getMaxOnGpu(const int* G_arr, int dataSize, int* Out) {
    int tid = threadIdx.x;
    int gthIdx = tid + blockIdx.x * blockSize;
    const int gridSize = blockSize * gridDim.x;
    int sum = 0;
    for (int i = gthIdx; i < dataSize; i += gridSize)
        sum += G_arr[i];
    __shared__ int particalMax[blockSize];
    particalMax[tid] = sum;
    __syncthreads();
    for (int stride = blockSize / 2; stride> 0; stride>>= 1) { 
        if (tid < stride)
            if (particalMax[tid] < particalMax[tid + stride])
                particalMax[tid] = particalMax[tid + stride];
        __syncthreads();
    }
    if (tid == 0)
        Out[blockIdx.x] = particalMax[0];
}

__global__ void kernelOutput(int* G_arr, int* Output, int dataSize) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < dataSize)
        G_arr[tid] = Output[tid];
    else
        return;
}

__host__ void Radix(int *C_arr,int *C_Output,int *Count,int dataSize, int digit) {

    for (int i = 1; i < 10; i++)
        Count[i] += Count[i - 1];

    for (int i = dataSize - 1; i >= 0; i--)
    {
        C_Output[Count[(C_arr[i] / digit) % 10] - 1] = C_arr[i];
        Count[(C_arr[i] / digit) % 10]--;
    }

}

void initializeArr(int* C_arr, int dataSize) {

    time_t t;
    srand((unsigned)time(&t));

    for (int i = 0; i < dataSize; i++)
        C_arr[i] = rand() % dataSize;

    printf("Array was constructed\n");

}

void printArr(int* C_arr, int dataSize) {

    for (int i = 0; i < dataSize; i++)
        printf("%d \n", C_arr[i]);

}

int main(int argc,char *argv[])
{   

    double runTime;
    int *C_arr, *G_arr, *Count,*C_Output, *Output,*G_Count;
    int* max, * GpuMax, reduceSize, dataSize = 1000;

    if (argc == 2) {
        dataSize = atoi(argv[1]);
    }

    clock_t gpuTime;
    size_t dataBytes = dataSize * sizeof(int);
    size_t countBytes = 10 * sizeof(int);

    dim3 threads(128);
    dim3 blocks(128);

    reduceSize = blocks.x;

    C_arr = (int*)malloc(dataBytes);
    C_Output = (int*)malloc(dataBytes);

    Count = (int*)malloc(countBytes);
    max = (int*)malloc(sizeof(int));

    hipMalloc((int**)&G_arr,dataBytes);
    hipMalloc((int**)&Output,dataBytes);

    hipMalloc((int**)&G_Count, countBytes);
    hipMalloc((int**)&GpuMax, sizeof(int));

    initializeArr(C_arr, dataSize);

    hipMemcpy(G_arr, C_arr, dataBytes, hipMemcpyHostToDevice);

    gpuTime = clock();

    hipDeviceSynchronize();

    getMaxOnGpu << <blocks, threads>> > (G_arr, dataSize, GpuMax);
   
    getMaxOnGpu<< <1, threads >> > (GpuMax, reduceSize, GpuMax);

    hipMemcpy(max, GpuMax, sizeof(int), hipMemcpyDeviceToHost);

    for (int digit = 1; *max / digit > 0; digit *= 10)
    {
        hipMemset(G_Count, 0, countBytes);
        kernelCount << < blocks, threads >> > (G_arr, G_Count, dataSize, digit);
        hipMemcpy(Count, G_Count, countBytes, hipMemcpyDeviceToHost);
        Radix(C_arr, C_Output, Count ,dataSize, digit);
        hipMemcpy(Output, C_Output, dataBytes, hipMemcpyHostToDevice);
        kernelOutput << <blocks, threads >> > (G_arr, Output, dataSize);
        hipMemcpy(C_arr, G_arr, dataBytes, hipMemcpyDeviceToHost);
    }
  
    gpuTime = clock() - gpuTime;
    
    runTime = ((double)gpuTime) / CLOCKS_PER_SEC;
    
    hipFree(G_arr);
    hipFree(Output);
    hipFree(G_Count);
    hipFree(GpuMax);

    printArr(C_arr, dataSize);

    free(C_arr);
    free(C_Output);
    free(Count);
    free(max);

    printf("%d dimensional array was sorted in %f seconds\n",dataSize, runTime);

    hipDeviceReset();

    return 0;
}
